#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "julia.h"
#include "graph.h"
#include <pshpack2.h>
int main()
{
//	BYTE graph[DIMX*DIMY*3];
	BYTE *graph = (BYTE*) malloc(sizeof(BYTE)*DIMX*DIMY * 3);
	unsigned char *dev_bitmap;
	hipMalloc((void**)&dev_bitmap, DIMX*DIMY * 3);
	hipMemset((void*)dev_bitmap, 255, DIMX*DIMY * 3);
	const dim3 grid(DIMX/32, 1);
	const dim3 thread(32, 1);
	kernel << <grid, thread >> >(dev_bitmap);
	hipMemcpy(graph, dev_bitmap, DIMX*DIMY * 3, hipMemcpyDeviceToHost);
	Snapshot((BYTE*)graph, DIMX, DIMY, "D:\\graph.bmp");
	free(graph);
	hipFree(dev_bitmap);
	return 0;
}

//void main()
//{
//	int i = 0, j = 0;
//	struct {
//		BYTE b;
//		BYTE g;
//		BYTE r;
//	} pRGB[240][320];  // ����λͼ����
//
//	memset(pRGB, 0, sizeof(pRGB)); // ���ñ���Ϊ��ɫ
//
//	// ���м仭һ��100*100�ľ���
//	for (i = 70; i<170; i++){
//		for (j = 110; j<210; j++){
//			pRGB[i][j].r = 0xff;
//		}
//	}
//
//	// ����BMPͼƬ
//	Snapshot((BYTE*)pRGB, 320, 240, "D:\\rgb.bmp");
//}
//
