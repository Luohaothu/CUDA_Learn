#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include <assert.h>

#define CUDA_CALL(x){	const hipError_t a = (x); \
						if (a != hipSuccess) \
												{\
							printf("\nCUDA Error: %s (err_num = %d) \n", hipGetErrorString(a), a);\
							hipDeviceReset(); assert(0); \
												}\
										}
#define KERNEL_LOOP 65536

typedef unsigned int u32;
__constant__ static const u32 const_data_01 = 0x55555555;
__constant__ static const u32 const_data_02 = 0x77777777;
__constant__ static const u32 const_data_03 = 0x33333333;
__constant__ static const u32 const_data_04 = 0x11111111;

__device__ static u32 data_01 = 0x55555555;
__device__ static u32 data_02 = 0x77777777;
__device__ static u32 data_03 = 0x33333333;
__device__ static u32 data_04 = 0x11111111;

__global__ void const_test_gpu_gmem(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < num_elements)
	{
		u32 d = 0x55555555;
		for (int i = 0; i < KERNEL_LOOP; i++)
		{
			d ^= data_01;
			d |= data_02;
			d &= data_03;
			d |= data_04;
		}

		data[tid] = d;
	}
}

__global__ void const_test_gpu_const(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < num_elements)
	{
		u32 d = const_data_01;
		for (int i = 0; i < KERNEL_LOOP; i++)
		{
			d ^= const_data_01;
			d |= const_data_02;
			d &= const_data_03;
			d |= const_data_04;
		}

		data[tid] = d;
	}
}

__host__ void wait_exit(void)
{
	char ch;
	printf("\nPress any key to exit\n");
	ch = getch();
}

__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if (hipPeekAtLastError() != hipSuccess)
	{
		printf("\n%s%s%s\n", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		wait_exit();
		exit(1);
	}
}

__host__ void gpu_kernel(void)
{
	const u32 num_elements = (128 * 1024);
	const u32 num_threads = 256;
	const u32 num_blocks = (num_elements + (num_threads - 1)) / num_threads;
	const u32 num_bytes = num_elements * sizeof(u32);
	int max_device_num;
	const int max_runs = 6;

	CUDA_CALL(hipGetDeviceCount(&max_device_num));

	for (int device_num = 0; device_num < max_device_num; device_num++)
	{
		CUDA_CALL(hipSetDevice(device_num));

		for (int num_test = 0; num_test < max_runs; num_test++)
		{
			u32 * data_gpu;
			hipEvent_t kernel_start1, kernel_stop1;
			hipEvent_t kernel_start2, kernel_stop2;
			float delta_time1 = 0.0f, delta_time2 = 0.0f;
			hipDeviceProp_t device_prop;
			char device_prefix[261];

			CUDA_CALL(hipMalloc(&data_gpu, num_bytes));
			CUDA_CALL(hipEventCreate(&kernel_start1));
			CUDA_CALL(hipEventCreate(&kernel_start2));
			CUDA_CALL(hipEventCreateWithFlags(&kernel_stop1, hipEventBlockingSync));
			CUDA_CALL(hipEventCreateWithFlags(&kernel_stop2, hipEventBlockingSync));

			CUDA_CALL(hipGetDeviceProperties(&device_prop, device_num));
			sprintf(device_prefix, "ID: %d Name: %s", device_num, device_prop.name);

			//Warm up run
			const_test_gpu_gmem << <num_blocks, num_threads >> >(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal startup kernel");

			//Do the literal kernel
			CUDA_CALL(hipEventRecord(kernel_start1, 0));
			const_test_gpu_gmem << <num_blocks, num_threads >> >(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal runtime kernel");

			CUDA_CALL(hipEventRecord(kernel_stop1, 0));
			CUDA_CALL(hipEventSynchronize(kernel_stop1));
			CUDA_CALL(hipEventElapsedTime(&delta_time1, kernel_start1, kernel_stop1));
			//printf("\nLiteral Elapsed time: %.3fms\n", delta_time1);

			//Warm up run
			const_test_gpu_const << <num_blocks, num_threads >> >(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from constant startup kernel");

			//Do the constant kernel
			CUDA_CALL(hipEventRecord(kernel_start2, 0));

			const_test_gpu_const << <num_blocks, num_threads >> >(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from constatn runtime kernel");

			CUDA_CALL(hipEventRecord(kernel_stop2, 0));
			CUDA_CALL(hipEventSynchronize(kernel_stop2));
			CUDA_CALL(hipEventElapsedTime(&delta_time2, kernel_start2, kernel_stop2));
			//printf("\nConst Elapsed time: %.3fms\n", delta_time2);

			if (delta_time1 > delta_time2)
			{
				printf("\n%s Constant version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",
					device_prefix, delta_time1 - delta_time2, delta_time1, delta_time2);
			}
			else
			{
				printf("\n%s Literal version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",
					device_prefix, delta_time2 - delta_time1, delta_time1, delta_time2);
			}

			CUDA_CALL(hipEventDestroy(kernel_start1));
			CUDA_CALL(hipEventDestroy(kernel_start2));
			CUDA_CALL(hipEventDestroy(kernel_stop1));
			CUDA_CALL(hipEventDestroy(kernel_stop2));
			CUDA_CALL(hipFree(data_gpu));
		}

		CUDA_CALL(hipDeviceReset());
		printf("\n");
	}

	wait_exit();
}

int main()
{
	gpu_kernel();
	return 0;
}