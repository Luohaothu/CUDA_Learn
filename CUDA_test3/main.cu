#include "kernel.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void)
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not handle overlaps, so no speed up from streams\n");
		return 0;
	}
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	hipMalloc((void**)&dev_a0, N * sizeof(int));
	hipMalloc((void**)&dev_b0, N * sizeof(int));
	hipMalloc((void**)&dev_c0, N * sizeof(int));

	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	for (int i = 0; i < FULL_DATA_SIZE; i += N * 2)
	{
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		kernel << <N / 256, 256, 0, stream0 >> >(dev_a0, dev_b0, dev_c0);

		hipMemcpyAsync(dev_b1, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel << <N / 256, 256, 0, stream1 >> >(dev_a1, dev_b1, dev_c1);
		
		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time taken: %3.1f ms\n", elapsedTime);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	return 0;
}